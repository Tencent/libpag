#include "hip/hip_runtime.h"
#ifndef SDK_INVERT_PROC_AMP
#	define SDK_INVERT_PROC_AMP

#	include "PrGPU/KernelSupport/KernelCore.h" //includes KernelWrapper.h
#	include "PrGPU/KernelSupport/KernelMemory.h"

#	if GF_DEVICE_TARGET_DEVICE
		GF_KERNEL_FUNCTION(ProcAmp2Kernel,
			((const GF_PTR(float4))(inSrc))
			((GF_PTR(float4))(outDst)),
			((int)(inSrcPitch))
			((int)(inDstPitch))
			((int)(in16f))
			((unsigned int)(inWidth))
			((unsigned int)(inHeight))
			((float)(inBrightness))
			((float)(inContrast))
			((float)(inHueCosSaturation))
			((float)(inHueSinSaturation)),
			((uint2)(inXY)(KERNEL_XY)))
		{
			if (inXY.x < inWidth && inXY.y < inHeight)
			{
				/* We'll operate in-place on 16f/32f data */
				float4 pixel = ReadFloat4(inSrc, inXY.y * inSrcPitch + inXY.x, !!in16f);

				/* RGB -> YUV */
				float srcY = pixel.z * 0.299f     + pixel.y * 0.587f     + pixel.x * 0.114f;
				float srcU = pixel.z * -0.168736f + pixel.y * -0.331264f + pixel.x * 0.5f;
				float srcV = pixel.z * 0.5f       + pixel.y * -0.418688f + pixel.x * -0.081312f;

				/* Render ProcAmp */
				float dstY = (inContrast * srcY) + inBrightness;
				float dstU = (srcU * inHueCosSaturation) + (srcV * -inHueSinSaturation);
				float dstV = (srcV * inHueCosSaturation) + (srcU *  inHueSinSaturation);

				/* YUV -> RGB */
				pixel.z = dstY * 1.0f + dstU * 0.0f       + dstV * 1.402f;
				pixel.y = dstY * 1.0f + dstU * -0.344136f + dstV * -0.714136f;
				pixel.x = dstY * 1.0f + dstU * 1.772f     + dstV * 0.0f;

				WriteFloat4(pixel, outDst, inXY.y * inDstPitch + inXY.x, !!in16f);
			}
		}

		GF_KERNEL_FUNCTION(InvertColorKernel,
			((const GF_PTR(float4))(inSrc))
			((GF_PTR(float4))(outDst)),
			((int)(inSrcPitch))
			((int)(inDstPitch))
			((int)(in16f))
			((unsigned int)(inWidth))
			((unsigned int)(inHeight)),
			((uint2)(inXY)(KERNEL_XY)))
		{
			if (inXY.x < inWidth && inXY.y < inHeight)
			{
				float4 pixel = ReadFloat4(inSrc, inXY.y * inSrcPitch + inXY.x, !!in16f);

				pixel.x = fmax(fmin(1.0f, pixel.x), 0.0f);
				pixel.y = fmax(fmin(1.0f, pixel.y), 0.0f);
				pixel.z = fmax(fmin(1.0f, pixel.z), 0.0f);
				pixel.w = fmax(fmin(1.0f, pixel.w), 0.0f);

				pixel.x = 1.0 - pixel.x;
				pixel.y = 1.0 - pixel.y;
				pixel.z = 1.0 - pixel.z;

				WriteFloat4(pixel, outDst, inXY.y * inDstPitch + inXY.x, !!in16f);
			}
		}
#	endif

#	if __NVCC__

		void Invert_Color_CUDA (
			float const *src,
			float *dst,
			unsigned int srcPitch,
			unsigned int dstPitch,
			int is16f,
			unsigned int width,
			unsigned int height)
		{
			dim3 blockDim(16, 16, 1);
			dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);

			InvertColorKernel <<< gridDim, blockDim, 0 >>> ((float4 const*)src, (float4*)dst, srcPitch, dstPitch, is16f, width, height);

			hipDeviceSynchronize();
		}

		void ProcAmp_CUDA (
			float const *src,
			float *dst,
			unsigned int srcPitch,
			unsigned int dstPitch,
			int	is16f,
			unsigned int width,
			unsigned int height,
			float inBrightness,
			float inContrast,
			float inHueCosSaturation,
			float inHueSinSaturation)
		{
			dim3 blockDim (16, 16, 1);
			dim3 gridDim ( (width + blockDim.x - 1)/ blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1 );

			ProcAmp2Kernel <<< gridDim, blockDim, 0 >>> ((float4 const*) src, (float4*) dst, srcPitch, dstPitch, is16f, width, height, inBrightness, inContrast, inHueCosSaturation, inHueSinSaturation );

			hipDeviceSynchronize();
		}
#	endif //GF_DEVICE_TARGET_HOST

#endif
